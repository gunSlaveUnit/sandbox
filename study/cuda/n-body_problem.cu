#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <iostream>
#include <fstream>

#include "hip/hip_runtime.h"
#include ""

#define n 1000
#define t0 0.0
#define tmax 10.0
#define tau 0.01
#define steps (tmax - t0) / tau
#define a1 1.0f
#define a2 2.0f
#define p1 3
#define p2 2
#define m 1
#define maxpos 5.0f
#define minpos -5.0f
#define blocksize 32
#define z 4

__device__ double sum(double* u, int i, int k, int f) {
	double sum = 0.0;
	for (int j = 0; j < n; ++j) {
		if (i == j) continue;
		double* ie = u + (k * n + i) * z;
		double* je = u + (k * n + j) * z;
		double xi = ie[2];
		double yi = ie[3];
		double xj = je[2];
		double yj = je[3];
		double dx = xj - xi;
		double dy = yj - yi;
		double d1 = a1 * (f ? dx : dy);
		double d2 = a2 * (f ? dx : dy);
		sum += d1 / pow(dx * dx + dy * dy, p1 / 2) -
			d2 / pow(dx * dx + dy * dy, p2 / 2);
	}
	return sum / m;
}

__global__ void Kernel(double* u) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= n) return;

	for (int k = 0; k < steps; ++k) {
		double* ce = u + (k * n + i) * z;
		double* ne = u + ((k + 1) * n + i) * z;
		ne[0] = ce[0] + tau * sum(u, i, k, 0); // vx
		ne[1] = ce[1] + tau * sum(u, i, k, 1); // vy
		ne[2] = ce[2] + tau * ce[0];           // x
		ne[3] = ce[3] + tau * ce[1];           // y

		if (!(minpos <= ne[2] && ne[2] <= maxpos)) { 
			ne[0] *= -1; 
			ne[2] += tau * ne[0];
		}
		if (!(minpos <= ne[3] && ne[3] <= maxpos)) { 
			ne[1] *= -1; 
			ne[3] += tau * ne[1];
		}

		__syncthreads();
	}
}

int main() {
	srand(time(0));
	int size = n * z * (steps + 1) * sizeof(double);
	double* u = (double*)malloc(size); // vx vy x y
	for (int i = 0; i < n; ++i) {
		u[i * z] = 0.0;
		u[i * z + 1] = 0.0;
		u[i * z + 2] = (double)rand() / RAND_MAX - 0.5;
		u[i * z + 3] = (double)rand() / RAND_MAX - 0.5;
	}

	double* udev = nullptr;
	hipMalloc((void**)&udev, size);
	hipMemcpy(udev, u, size, hipMemcpyHostToDevice);

	float time;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start, NULL);
	Kernel << <(n / blocksize) + 1, blocksize >> > (udev);
	hipEventRecord(stop, NULL);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);

	hipMemcpy(u, udev, size, hipMemcpyDeviceToHost);

	std::ofstream f("results.txt");

	for (int k = 0; k <= steps; ++k) {
		for (int i = 0; i < n; ++i) {
			f << u[(k * n + i) * z + 2] << " ";
			f << u[(k * n + i) * z + 3] << std::endl;
		}
	}

	f.flush();
	f.close();

	std::cout << "Done for " << time / 1000.0 << "s";

	hipFree(udev);
	free(u);
}
