#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include <memory>

#include "hip/hip_runtime.h"
#include ""

__global__ void Kernel(double* ud, double* oldud, double* newud, int n, int m, double r) {
	int i = blockIdx.x * blockDim.x + threadIdx.x + 1;
	int j = blockIdx.y * blockDim.y + threadIdx.y + 1;

	if (i >= n - 1 || j >= m - 1) return;

	newud[i * m + j] = 2.0 * ud[i * m + j] - oldud[i * m + j] +
		r * (ud[(i - 1) * m + j] +
			ud[(i + 1) * m + j] +
			ud[i * m + j - 1] +
			ud[i * m + j + 1] -
			4.0 * ud[i * m + j]);

	__syncthreads();

	oldud[i * m + j] = ud[i * m + j];
	ud[i * m + j] = newud[i * m + j];
}

int main() {
	const int BLOCKSIZE = 32;
	const double X = 1.0, Y = 1.3, v0 = 2.4,
		xmax = 1.5, ymax = 2.0, h = 0.02,
		t0 = 0.0, tmax = 1.0, tau = 0.01,
		c = 1.0;
	const int n = int((xmax / h) + 1), m = int((ymax / h) + 1);
	const double r = c * c * tau * tau / (h * h);

	int size = n * m * sizeof(double);

	double* oldu = (double*)malloc(size);
	double* u = (double*)malloc(size);

	int i = int(X / h), j = int(Y / h);
	memset(u, 0, size);
	memset(oldu, 0, size);
	oldu[i * m + j] = -tau * v0;

	double* oldud = nullptr;
	double* ud = nullptr;
	double* newud = nullptr;
	hipMalloc((void**)&oldud, size);
	hipMalloc((void**)&ud, size);
	hipMalloc((void**)&newud, size);
	hipMemcpy(oldud, oldu, size, hipMemcpyHostToDevice);
	hipMemcpy(ud, u, size, hipMemcpyHostToDevice);

	float time;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start, NULL);
	for (float t = t0; t < tmax; t += tau) {
		Kernel << <dim3(n / BLOCKSIZE + 1, m / BLOCKSIZE + 1),
			dim3(BLOCKSIZE, BLOCKSIZE) >> > (ud, oldud, newud, n, m, r);
		hipDeviceSynchronize();
	}
	hipEventRecord(stop, NULL);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);

	hipMemcpy(u, ud, size, hipMemcpyDeviceToHost);

	FILE* f;
	fopen_s(&f, "FINAL.txt", "w");
	if (f) {
		for (i = 0; i < n; ++i) {
			for (j = 0; j < m; ++j)
				fprintf(f, "%8.10f\t ", u[i * m + j]);
			fprintf(f, "\n");
		}
		fclose(f);
	}

	printf("Done for %fs", time / 1000);

	hipEventDestroy(start);
	hipEventDestroy(stop);
	hipFree(oldud);
	hipFree(ud);
	hipFree(newud);
	free(oldu);
	free(u);
}
