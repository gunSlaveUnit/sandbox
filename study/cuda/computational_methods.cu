#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <string.h>

#include "hip/hip_runtime.h"
#include ""

#define n 2
#define t0 0.0
#define tmax 10.0
#define tau 0.001
#define amountMethods 5

const int steps = (tmax - t0) / tau;

__device__ double f1(double* y, double t) {
	return y[1];
}

__device__ double f2(double* y, double t) {
	return -0.19 * t * y[1] - t * t * y[0];
}

__device__ double (*fs[n])(double*, double) = { f1, f2 };

__device__ void KernelEuler(double* y, int k, double t) {
	int i = threadIdx.x;
	y[(k + 1) * n + i] = y[k * n + i] + tau * fs[i](y + k * n, t);
	__syncthreads();
}

__device__ void KernelRungeKuttaTwo(double* y, int k, double t) {
	__shared__ double yy[n], ff[n];
	for (int i = 0; i < n; ++i) {
		yy[i] = y[k * n + i] + 0.5 * tau * fs[i](y + k * n, t);
		ff[i] = fs[i](yy, t + 0.5 * tau);
		y[(k + 1) * n + i] = y[k * n + i] + tau * ff[i];
	}
}

__device__ void KernelForecastCorrection(double* y, int k, double t) {
	__shared__ double ff[n], ffk[n];
	for (int i = 0; i < n; ++i) {
		ffk[i] = fs[i](y + k * n, t);
		y[(k + 1) * n + i] = y[k * n + i] + tau * fs[i](y + k * n, t);
		ff[i] = fs[i](y + (k + 1) * n + i, t + tau);
		y[k * n + i] += tau * (ff[i] + ffk[i]) / 2.0;
	}
}

__device__ void KernelRungeKuttaFour(double* y, int k, double t) {
	const int amountCoefs = 4;
	__shared__ double yy[n], r[n][amountCoefs];
	for (int i = 0; i < n; ++i) {
		r[i][0] = tau * fs[i](y + k * n, t);
		yy[i] = y[k * n + i] + 0.5 * r[i][0];
		r[i][1] = tau * fs[i](yy, t + 0.5 * tau);
		yy[i] = y[k * n + i] + 0.5 * r[i][1];
		r[i][2] = tau * fs[i](yy, t + 0.5 * tau);
		yy[i] = y[k * n + i] + 0.5 * r[i][2];
		r[i][3] = tau * fs[i](yy, t + 0.5 * tau);
		yy[i] = y[k * n + i] + 0.5 * r[i][3];
		y[(k + 1) * n + i] += (r[i][0] + 2 * r[i][1] + 2 * r[i][2] + r[i][3]) / 6.0;
	}
}

__device__ double partialDerivative(double (*f)(double*, double), double* y, double t, int j) {
	__shared__ double yy[n];
	double h = 0.001;
	for (int i = 0; i < n; ++i) yy[i] = y[i];
	yy[j] += h;
	return (f(yy, t) - f(y, t)) / h;
}

__device__ void KernelImplicitEuler(double* y, int k, double t) {
	double p[n] = {}, a[n][n] = {}, f[n] = {};
	double determinants[n + 1] = {};
	for (int i = 0; i < n; ++i)
		for (int j = 0; j < n; ++j) {
			a[i][j] = partialDerivative(fs[i], y, t, j);
			if (i == j) a[i][j] -= 1 / tau;
		}
	for (int i = 0; i < n; ++i) f[i] = -fs[i](y + k * n, t);
	determinants[0] = a[0][0] * a[1][1] - a[0][1] * a[1][0];
	determinants[1] = a[1][1] * f[0] - f[1] * a[0][1];
	determinants[2] = a[0][0] * f[1] - f[0] * a[1][0];
	for (int i = 0; i < n; ++i) {
		p[i] = determinants[i + 1] / determinants[0];
		y[(k + 1) * n + i] = y[k * n + i] + p[i];
	}
}

__device__ void (*methods[amountMethods])(double*, int, double) = { KernelEuler, KernelRungeKuttaTwo,
		KernelForecastCorrection, KernelRungeKuttaFour, KernelImplicitEuler };

__global__ void Compute(int m, double* ydev) {
	for (int k = 0; k <= steps; ++k) {
		double t = k * tau + t0;
		methods[m](ydev, k, t);
	}
}

char* concat(const char* s1, const char* s2)
{
	const size_t len1 = strlen(s1);
	const size_t len2 = strlen(s2);
	char* result = (char*)malloc(len1 + len2 + 1);
	memcpy(result, s1, len1);
	memcpy(result + len1, s2, len2 + 1); 
	return result;
}

int main() {
	char* fileExt = ".csv";
	char* titlesMethods[amountMethods] = { "Explicit Euler", "Runge-Kutta 2",
		"Forecast-Correction", "Runge-Kutta 4", "Implicit Euler" };

	int size = n * (steps + 1) * sizeof(double);
	double* y = (double*)malloc(size);
	y[0] = 0.0;
	y[1] = 0.05;
	float time;

	double* yDev = nullptr;
	hipMalloc((void**)&yDev, size);

	hipMemcpy(yDev, y, n * sizeof(double), hipMemcpyHostToDevice);

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	for (int m = 0; m < amountMethods; ++m) {
		double averageTime = 0.0;
		int amountIters = 5;

		printf("[%s Method]\n", titlesMethods[m]);

		for (int i = 0; i < amountIters; ++i) {
			hipEventRecord(start, NULL);

			Compute << <1, n >> > (m, yDev);

			hipEventRecord(stop, NULL);
			hipEventSynchronize(stop);
			hipEventElapsedTime(&time, start, stop);
			hipMemcpy(y, yDev, size, hipMemcpyDeviceToHost);
			printf("| Iter #%d Done <%fs> |\n", i, time / 1000.0);
			averageTime += time / 1000.0;
		}
		printf("* Average time: <%fs> *\n\n", averageTime / amountIters);
		FILE* f = fopen(concat(titlesMethods[m], fileExt), "w");
		for (int k = 0; k <= steps; ++k) {
			fprintf(f, "%f, %f, %f\n", k * tau + t0, y[k * n], y[k * n + 1]);
		}
		fflush(f);
		fclose(f);
	}	
	
	hipEventDestroy(start);
	hipEventDestroy(stop);

	hipFree(yDev);
	free(y);
}
